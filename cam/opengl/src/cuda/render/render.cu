
#include <hip/hip_runtime.h>
#include <math.h>
#define uint8 unsigned char
#define uint16 unsigned short int

__global__ void _u162g8_kernel(void *d_data, void *d_tex, int size, int thres, int vmin, int vmax)
{
    uint8 data8;
    //uint8 r,g;
    uint16 data16, data16_scaled;
    uint16 *data_p=(uint16 *)d_data;
    uint8 *tex_p=(uint8 *)d_tex;

    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    //uint16 factor = (uint16)(4096./(vmax-vmin));
      
      
    data16=data_p[idx];


    if(idx<size){
      if (data16>vmax){
        data16_scaled=4095;
      } else if (data16 < vmin){
        data16_scaled=0;
      } else {
        data16_scaled=(data16-vmin)*4096/(vmax-vmin);
      }

      data8=(uint8)(data16_scaled/16);

      if (data16==4095){ 
        tex_p[3*idx]=255;
        tex_p[3*idx+1]=0;
        tex_p[3*idx+2]=0;
      }else if(data16<thres){
        tex_p[3*idx]=data8*.75;
        tex_p[3*idx+1]=data8*.75;
        tex_p[3*idx+2]=data8;
      }else{
        tex_p[3*idx]=data8;
        tex_p[3*idx+1]=data8;
        tex_p[3*idx+2]=data8;
      }
    }
}

__global__ void _u82g8_kernel(void *d_data, void *d_tex, int size, int thres, int vmin, int vmax)
{
    uint8 data8, data8_scaled;
    //uint16 data16, data16_scaled;
    uint8 *data_p=(uint8 *)d_data;
    uint8 *tex_p=(uint8 *)d_tex;

    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    //uint16 factor = (uint16)(4096./(vmax-vmin));
      
      
    data8=data_p[idx];


    if(idx<size){
      if (data8>vmax){
        data8_scaled=255;
      } else if (data8 < vmin){
        data8_scaled=0;
      } else {
        data8_scaled=(data8-vmin)*255/(vmax-vmin);
      }

      if (data8==255){ 
        tex_p[3*idx]=255;
        tex_p[3*idx+1]=0;
        tex_p[3*idx+2]=0;
      }else if(data8<thres){
        tex_p[3*idx]=data8_scaled*.75;
        tex_p[3*idx+1]=data8_scaled*.75;
        tex_p[3*idx+2]=data8_scaled;
      }else{
        tex_p[3*idx]=data8_scaled;
        tex_p[3*idx+1]=data8_scaled;
        tex_p[3*idx+2]=data8_scaled;
      }
    }
}

__global__ void _u82rgb8_kernel(void *d_data, void *d_tex, int size, int w, int thres, int vmin, int vmax)
{
    float datafg;   
    uint8 data8r,data8g,data8b;
    uint8 *data_p=(uint8 *)d_data;
    uint8 *tex_p=(uint8 *)d_tex;

    int idx = blockIdx.x*blockDim.x+threadIdx.x; 
    int idrow=(idx/(w/2));
    int idcol=(idx%(w/2));
    int idxrgb=idcol*2+idrow*w*2;

    if (idxrgb+w+1<size)
    {
        /*data8r=data_p[idxrgb];
        datafg=(float)data_p[idxrgb+1];
        datafg+=(float)data_p[idxrgb+w];
        data8b=data_p[idxrgb+w+1];*/
        datafg=(float)data_p[idxrgb];
        datafg+=(float)data_p[idxrgb+w+1];
        datafg*=0.4;
        data8r=data_p[idxrgb+w];
        data8b=data_p[idxrgb+1];

        if (data8r>vmax) {data8r=255;} 
        else if (data8r<vmin) {data8r=0;} 
        else {data8r=(data8r-vmin)*255/(vmax-vmin);}

        if (data8b>vmax) {data8b=255;} 
        else if (data8b<vmin) {data8b=0;} 
        else {data8b=(data8b-vmin)*255/(vmax-vmin);}

        if (datafg>vmax) {data8g=255;} 
        else if (datafg<vmin) {data8g=0;} 
        else {data8g=(datafg-vmin)*255/(vmax-vmin);}

        tex_p[3*(idxrgb)    ]=data8r;
        tex_p[3*(idxrgb)    +1]=data8g;
        tex_p[3*(idxrgb)    +2]=data8b;

        tex_p[3*(idxrgb+1)  ]=data8r;
        tex_p[3*(idxrgb+1)  +1]=data8g;
        tex_p[3*(idxrgb+1)  +2]=data8b;

        tex_p[3*(idxrgb+w)  ]=data8r;
        tex_p[3*(idxrgb+w)  +1]=data8g;
        tex_p[3*(idxrgb+w)  +2]=data8b;

        tex_p[3*(idxrgb+w+1)]=data8r;
        tex_p[3*(idxrgb+w+1)+1]=data8g;
        tex_p[3*(idxrgb+w+1)+2]=data8b;
    }
}


/*
__global__ void _u162f32_kernel(void *d_raw, void *d_preproc, int size)
{
    uint16 *raw_p=(uint16 *)d_raw;
    float32 *preproc_p=(float32 *)d_preproc;
 
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
      
    if(idx<size){
      preproc_p[idx]=(float32)raw_p[idx];
    }
}

extern "C"  void uint16_to_float32(void *d_raw, void *d_preproc, int size)
{
  _u162f32_kernel<<<10020,256>>>(d_raw, d_preproc, size);
}


extern "C"  void float32_to_gray8(void *d_data, void *d_tex, int size, int thres, int vmin, int vmax)
{
  _f322g8_kernel<<<10020,256>>>(d_data, d_tex, size, thres, vmin, vmax, INVERTED);
}
*/

extern "C"  void uint16_to_gray8(void *d_data, void *d_tex, int size, int thres, int vmin, int vmax)
{
  int numBlocks=(int)(ceil(size/256.));
  _u162g8_kernel<<<numBlocks,256>>>(d_data, d_tex, size, thres, vmin, vmax);
}

extern "C"  void uint8_to_gray8(void *d_data, void *d_tex, int size, int thres, int vmin, int vmax)
{
  int numBlocks=(int)(ceil(size/256.));
  _u82g8_kernel<<<numBlocks,256>>>(d_data, d_tex, size, thres, vmin, vmax);
}

extern "C"  void uint8_to_rgb8(void *d_data, void *d_tex, int size, int w, int thres, int vmin, int vmax)
{
  int numBlocks=(int)(ceil(size/4/256.));//each thread takes 4 pixels
  _u82rgb8_kernel<<<numBlocks,256>>>(d_data, d_tex, size, w, thres, vmin, vmax);
}

/*extern "C"  void gray8(void *d_data, void *d_tex, int x, int y, int w, int h, uint8 thres, int vmin, int vmax)
{
  _gray8_kernel<<<10020,256>>>(d_data, d_tex, x, y, w, h, thres, vmin, vmax);
}*/

