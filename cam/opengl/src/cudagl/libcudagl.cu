#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#define MAXNUMRESOURCES 100

#  define SAFE_CALL(call) {                                                 \
    hipError_t err = call;                                                   \
    if( hipSuccess != err) {                                               \
        sprintf(errMsg, "Cuda error in file '%s' in line %i : %s.\n",       \
                __FILE__, __LINE__, hipGetErrorString(err) );              \
        return 1;                                                            \
    } }




//Global vars
struct hipGraphicsResource *cuda_pbo[MAXNUMRESOURCES];
int next_resource_idx=0;
int INVERTED=0;

//also works if a skip call to initCuda ???
extern "C" int initCuda(char *errMsg)
{
  int device;
  SAFE_CALL(hipGetDevice(&device));
  SAFE_CALL(cudaGLSetGLDevice(device));
  //hipSetDeviceFlags(hipDeviceMapHost);
  return 0;
}

//also works if a skip call to initCuda ???
extern "C" int makeCurrent(char *errMsg)
{
  int device;
  SAFE_CALL(hipGetDevice(&device));
  SAFE_CALL(cudaGLSetGLDevice(device));
  //hipSetDeviceFlags(hipDeviceMapHost);
  return 0;
}


extern "C" int regbuf(GLuint pbo, int *resource_idx, char *errMsg){
  if (next_resource_idx >= MAXNUMRESOURCES){
    sprintf(errMsg, "Maximum number of resources exceeded");
    return 1;
  }
  *resource_idx=next_resource_idx;
  next_resource_idx+=1;

  SAFE_CALL(hipGraphicsGLRegisterBuffer(&cuda_pbo[*resource_idx], pbo, cudaGraphicsMapFlagsNone));
  return 0;
}

extern "C"  int glmap(GLuint pbo, int resource_idx, void **d_ptr, char *errMsg)
{
  size_t num_bytes;
  SAFE_CALL(hipGraphicsMapResources(1, &cuda_pbo[resource_idx], 0));
  SAFE_CALL(hipGraphicsResourceGetMappedPointer(d_ptr, &num_bytes, cuda_pbo[resource_idx]));
  return 0;
}
  
extern "C"  int glunmap(GLuint pbo, int resource_idx, char *errMsg)
{
  SAFE_CALL(hipGraphicsUnmapResources(1, &cuda_pbo[resource_idx], 0));
  return 0;
}
